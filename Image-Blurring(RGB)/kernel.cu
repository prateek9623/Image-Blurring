#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "time.h";
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <opencv2/opencv.hpp>

const int RADIUS = 1;
const char* inputImageName = "input.jpg";
const char* outputImageName = "output.jpg";

using namespace cv;
using namespace std;


void showImage(Mat img, char *title) {
	namedWindow(title, CV_WINDOW_AUTOSIZE);
	imshow(title, img);
	waitKey(0);
}

__device__ void memSetSharedMem(int x, int y, uchar3 *sharedData, uchar3 *globalData, int maxX, int maxY) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		for (int posY = 0; posY <= RADIUS; posY++)
			for (int posX = 0; posX <= RADIUS; posX++) {
				if (x <= 0 && y <= 0) {
					sharedData[posX + posY * (blockDim.x + 2 * RADIUS)] = globalData[x+posX + maxX * (y+posY)];
				}
				else {
					sharedData[posX + posY * (blockDim.x + 2 * RADIUS)] = globalData[(x - RADIUS + posX) + maxX * (y - RADIUS + posY)];
				}
			}
	}
	else if (threadIdx.y == 0 && threadIdx.x == blockDim.x - 1) {
		for (int posY = 0; posY <= RADIUS; posY++)
			for (int posX = 0; posX <= RADIUS; posX++) {
				if (y <= 0) {
					sharedData[posX + threadIdx.x + RADIUS + posY * (blockDim.x + 2 * RADIUS)] = globalData[(x-posX) + maxX * (y+posY)];
				}
				else {
					sharedData[posX + threadIdx.x + RADIUS + posY * (blockDim.x + 2 * RADIUS)] = globalData[(x + posX) + maxX * (y - RADIUS + posY)];
				}
			}
	}
	else if (threadIdx.y == blockDim.y - 1 && threadIdx.x == 0) {
		for (int posY = 0; posY <= RADIUS; posY++)
			for (int posX = 0; posX <= RADIUS; posX++) {
				if (x <= 0) {
					sharedData[posX + (posY + RADIUS + threadIdx.y) * (blockDim.x + 2 * RADIUS)] = globalData[(x+posX) + maxX * (y+posY)];
				}
				else {
					sharedData[posX + (posY + RADIUS + threadIdx.y) * (blockDim.x + 2 * RADIUS)] = globalData[(x - RADIUS + posX) + maxX * (y + posY)];
				}
			}
	}
	else if (threadIdx.y == blockDim.y - 1 && threadIdx.x == blockDim.x - 1) {
		for (int posY = 0; posY <= RADIUS; posY++)
			for (int posX = 0; posX <= RADIUS; posX++) {
				if (x == maxX - 1 && y == maxY - 1) {
					sharedData[posX + RADIUS + threadIdx.x + (posY + RADIUS + threadIdx.y) * (blockDim.x + 2 * RADIUS)] = globalData[(x-posX) + maxX * (y-posY)];
				}
				else {
					sharedData[posX + RADIUS + threadIdx.x + (posY + RADIUS + threadIdx.y) * (blockDim.x + 2 * RADIUS)] = globalData[(x + posX) + maxX * (y + posY)];
				}
			}
	}
	else if (threadIdx.x == 0) {
		int posY = threadIdx.y + RADIUS;
		for (int posX = 0; posX <= RADIUS; posX++) {
			if (x <= 0) {
				sharedData[posX + posY * (blockDim.x + 2 * RADIUS)] = globalData[x+posX + maxX * y];
			}
			else {
				sharedData[posX + posY * (blockDim.x + 2 * RADIUS)] = globalData[(x - RADIUS + posX) + maxX * y];
			}

		}
	}
	else if (threadIdx.y == 0) {
		int posX = threadIdx.x + RADIUS;
		for (int posY = 0; posY <= RADIUS; posY++) {
			if (y <= 0) {
				sharedData[posX + posY * (blockDim.x + 2 * RADIUS)] = globalData[x + maxX * (y+posY)];
			}
			else {
				sharedData[posX + posY * (blockDim.x + 2 * RADIUS)] = globalData[x + maxX * (y - RADIUS + posY)];
			}

		}
	}
	else if (threadIdx.x == blockDim.x - 1) {
		int posY = threadIdx.y + RADIUS;
		for (int posX = 0; posX <= RADIUS; posX++) {
			if (x == maxX) {
				sharedData[posX + RADIUS + threadIdx.x + posY * (blockDim.x + 2 * RADIUS)] = globalData[x-posX + maxX * y];
			}
			else {
				sharedData[posX + RADIUS + threadIdx.x + posY * (blockDim.x + 2 * RADIUS)] = globalData[(x + posX) + maxX * y];
			}
		}
	}
	else if (threadIdx.y == blockDim.y - 1) {
		int posX = threadIdx.x + RADIUS;
		for (int posY = 0; posY <= RADIUS; posY++) {
			if (y == maxY - 1) {
				sharedData[posX + (posY + RADIUS + threadIdx.y) * (blockDim.x + 2 * RADIUS)] = globalData[x + maxX * (y-posY)];
			}
			sharedData[posX + (posY + RADIUS + threadIdx.y) * (blockDim.x + 2 * RADIUS)] = globalData[x + maxX * (y + posY)];
		}
	}
	else {
		sharedData[threadIdx.x + RADIUS + (threadIdx.y + RADIUS) * (blockDim.x + 2 * RADIUS)] = globalData[x + maxX * y];
	}
}

__global__ void findAverage(uchar3 *matrix, uchar3 *avgMatrix, int maxX, int maxY, int count) {
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	extern __shared__ uchar3 sharedData[];



	if (x < maxX  && y < maxY) {
		memSetSharedMem(x, y, sharedData, matrix, maxX, maxY);
		__syncthreads();
		int3 sum { 0,0,0 };;
		int sharedMaxX = blockDim.x + 2 * RADIUS;
		//if (threadIdx.x == 0 && blockIdx.x == 3 && threadIdx.y == 2 && blockIdx.y == 3)
		{
			for (int r = 0; r < 2 * RADIUS + 1; r++) {
				for (int c = 0; c < 2 * RADIUS + 1; c++) {
					//printf("%d ", sharedData[(threadIdx.x + c) + (sharedMaxX)*(threadIdx.y + r)]);
					sum.x += sharedData[(threadIdx.x + c) + (sharedMaxX)*(threadIdx.y + r)].x;
					sum.y += sharedData[(threadIdx.x + c) + (sharedMaxX)*(threadIdx.y + r)].y;
					sum.z += sharedData[(threadIdx.x + c) + (sharedMaxX)*(threadIdx.y + r)].z;
				}
				//printf("\n");
			}
			avgMatrix[x + maxX * y].x = (uchar)(sum.x / count);
			avgMatrix[x + maxX * y].y =(uchar) (sum.y / count);
			avgMatrix[x + maxX * y].z =(uchar) (sum.z / count);
		}
	}

}
//__global__ void findAverage(const uchar3 *matrix, uchar3 *avgMatrix, int maxX, int maxY,  int count) {
//	int x = threadIdx.x + blockIdx.x*blockDim.x;
//	int y = threadIdx.y + blockIdx.y*blockDim.y;
//	int index = x + maxX*y;
//	
//	if (x < maxX && y < maxY) 
//	{
//		int3 sum = { 0,0,0 };
//		for (int offsetY = y - RADIUS; offsetY <= y + RADIUS && offsetY < maxY; offsetY++) {
//			for (int offsetX = x - RADIUS; offsetX <= x + RADIUS && offsetX < maxX; offsetX++) {
//				if (offsetX >= 0 && offsetY >= 0)
//				{
//					int indexOffset = offsetY * maxX + offsetX;
//					sum.x += (int)matrix[indexOffset].x;
//					sum.y += (int)matrix[indexOffset].y;
//					sum.z += (int)matrix[indexOffset].z;
//				}
//			}
//		}
//		avgMatrix[index].x =  (uchar)(sum.x/count);
//		avgMatrix[index].y = (uchar)(sum.y / count);
//		avgMatrix[index].z = (uchar)(sum.z / count);
//		//matrix[index];
//		//__syncthreads();
//		//printf("%d ", avgMatrix[index]);
//	}
//}


int main()
{
	hipFree(0);
	Mat inputImage = imread(inputImageName, CV_LOAD_IMAGE_UNCHANGED);
	Mat outputImage = imread(inputImageName, CV_LOAD_IMAGE_UNCHANGED);

	
	if (inputImage.empty() || outputImage.empty()) {
		cerr << "Couldn't open file::" << inputImageName;
		exit(1);
	}

	uchar3* inputImageData = (uchar3*)inputImage.ptr<uchar3>(0);
	uchar3* outputImageData = (uchar3*)outputImage.ptr<uchar3>(0);

	uchar3* dInputImageData;
	uchar3* dOutputImageData;

	if (hipMalloc(&dInputImageData, sizeof(uchar3)*inputImage.rows*inputImage.cols) != hipSuccess) {
		cerr << "Couldn't allocate memory for input image";
		hipFree(dInputImageData);
		exit(1);
	};

	if (hipMalloc(&dOutputImageData, sizeof(uchar3)*inputImage.rows*inputImage.cols) != hipSuccess) {
		cerr << "Couldn't allocate memory for output image";
		hipFree(dOutputImageData);
		hipFree(dInputImageData);
		exit(1);
	};

	if (hipMemcpy(dInputImageData, inputImageData, sizeof(uchar3)*inputImage.rows*inputImage.cols, hipMemcpyHostToDevice) != hipSuccess) {
		cerr << "Couldn,t initialiZe device for input image";
		hipFree(dOutputImageData);
		hipFree(dInputImageData);
		exit(1);
	}

	if (hipMemset(dOutputImageData, 0, sizeof(uchar3)*inputImage.rows*inputImage.cols) != hipSuccess) {
		cerr << "Couldn,t initialiZe device Average Matrix";
		hipFree(dOutputImageData);
		hipFree(dInputImageData);
		exit(1);
	}
	const dim3 blockSize(32, 32, 1);
	const dim3 gridSize((inputImage.cols + blockSize.x - 1) / blockSize.x, (inputImage.rows + blockSize.y - 1) / blockSize.y, 1);
	int count = (RADIUS * 2 + 1)*(RADIUS * 2 + 1);

	int sharedMemSpace = (blockSize.x + 2 * RADIUS)*(blockSize.y + 2 * RADIUS);
	//printf("%d %d %d %d %d %d %d %d", blockSize.x, blockSize.y, gridSize.x, gridSize.y, inputImage.rows, inputImage.cols, outputImage.rows, outputImage.cols);
	findAverage <<<gridSize, blockSize, sharedMemSpace * sizeof(uchar3) >>> (dInputImageData, dOutputImageData, inputImage.cols, inputImage.rows, count);
	//findAverage <<<gridSize, blockSize >>> (dInputImageData, dOutputImageData, inputImage.cols, inputImage.rows, count);

	hipDeviceSynchronize();

	if (hipGetLastError() != hipSuccess) {
		cerr << "kernel launch failed: " << hipGetErrorString(hipGetLastError());
		hipFree(dOutputImageData);
		hipFree(dInputImageData);
		exit(1);
	}

	if (hipMemcpy(outputImageData, dOutputImageData, sizeof(uchar3)*inputImage.rows*inputImage.cols, hipMemcpyDeviceToHost) != hipSuccess) {
		cerr << "Couldn't copy original matrix memory from device to host";
		hipFree(dOutputImageData);
		hipFree(dInputImageData);
		exit(1);
	}
	hipFree(dInputImageData);
	hipFree(dOutputImageData);

	imwrite(outputImageName, outputImage);
	//showImage(inputImage, "Original Image ");
	//showImage(outputImage, "Blur Image");
	//delete[] inputImageData;
	//delete[] outputImageData;
	return 0;
}
